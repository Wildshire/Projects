/*
This is the function you need to implement. Quick reference:
- input rows: 0 <= y < ny
- input columns: 0 <= x < nx
- element at row y and column x is stored in data[x + y*nx]
- correlation between rows i and row j has to be stored in result[i + j*ny]
- only parts with 0 <= j <= i < ny need to be filled
*/

//THIS IS MY VERSION OF CP2B, TOOK PREPROCESSING PART

//Imports
#include <stdio.h>
#include <vector>
#include <math.h>
#include <typeinfo>
#include <algorithm>

#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>

//GPU auxiliary functions
static inline void check(hipError_t err, const char* context) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << context << ": "
            << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

#define CHECK(x) check(x, #x)

static inline int divup(int a, int b) {
    return (a + b - 1)/b;
}

//Not needed yet
//static inline int roundup(int a, int b) {
  //  return divup(a, b) * b;
//}
 

__global__ void mykernel(float* x, float* result, int nx, int ny) {
	int row_aux = threadIdx.x + blockIdx.x * blockDim.x;
    	int row = threadIdx.y + blockIdx.y * blockDim.y;
    	if (row >= ny || row_aux >= ny){
        	return;
	}
	//printf("Row: %d, Row_aux: %d, NX: %d, NY: %d\n", row,row_aux,nx,ny);
    	float pearson_coeff = 0.0;
    	for (int column = 0; column < nx; ++column) {
        	float first = x[row*nx + column];
        	float second = x[row_aux*nx + column];
		//printf("First: %f, Second: %f\n",first,second);
		pearson_coeff+=first*second;
    	}
    	result[row+row_aux*ny] = pearson_coeff;
	result[row_aux+row*ny] = pearson_coeff;
}

//CPU auxiliary functions
//Struct for sum
struct Sum{
	auto operator() (float n){ sum +=n; }
	float sum{0};
};


//Struc for square_sum
struct Square_sum{
	auto operator() (float n){ sq_sum += pow(n,2.0f); }
	float sq_sum{0};
};

void correlate(int ny, int nx, const float *data, float *result) {
	//asm("CPU PART I");
	//Auxiliary variables
	std::vector <float> x(ny*nx,0.0); //x matrix

	//Now x will cointain data mean and std normalized
	#pragma omp parallel for schedule(dynamic,1)
	for(int row=0;row<ny;++row){
		for (int column=0;column<nx;++column){
			x[column + row*nx]=data[column + row*nx];
		}
		//Mean
		//Sum all values
		Sum s = std::for_each(x.begin()+(row*nx),x.begin()+(nx+row*nx),Sum());
		//Substract mean to all of them
		std::for_each(x.begin()+(row*nx),x.begin()+(nx+row*nx),[&] (float &n){n=n-(s.sum/nx);});

		//Norm
		Square_sum sq_s = std::for_each(x.begin()+(row*nx),x.begin()+(nx+row*nx),Square_sum());
		//Divide norm
		std::for_each(x.begin()+(row*nx),x.begin()+(nx+row*nx),[&] (float &n){n=n/sqrt(sq_s.sq_sum);});
	}

	//asm("GPU PART");
	//Memory storing
	float* xGPU = NULL;
    	CHECK(hipMalloc((void**)&xGPU, ny * nx * sizeof(float)));
    	float* resultGPU = NULL;
    	CHECK(hipMalloc((void**)&resultGPU, ny * ny * sizeof(float)));
    	CHECK(hipMemcpy(xGPU, &*x.begin(), ny * nx * sizeof(float), hipMemcpyHostToDevice));

    	// Run kernel
    	dim3 dimBlock(16, 16);
    	dim3 dimGrid(divup(ny, dimBlock.x), divup(ny, dimBlock.y));
    	mykernel<<<dimGrid, dimBlock>>>(xGPU, resultGPU, nx, ny);
    	CHECK(hipGetLastError());

    	// Copy data back to CPU & release memory
    	CHECK(hipMemcpy(result, resultGPU, ny * ny * sizeof(float), hipMemcpyDeviceToHost));
    	CHECK(hipFree(xGPU));
    	CHECK(hipFree(resultGPU));
}
