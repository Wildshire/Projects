/*
This is the function you need to implement. Quick reference:
- input rows: 0 <= y < ny
- input columns: 0 <= x < nx
- element at row y and column x is stored in data[x + y*nx]
- correlation between rows i and row j has to be stored in result[i + j*ny]
- only parts with 0 <= j <= i < ny need to be filled
*/

//THIS IS MY VERSION OF CP2B, TOOK PREPROCESSING PART

//Imports
#include <stdio.h>
#include <vector>
#include <math.h>
#include <typeinfo>
#include <algorithm>

#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>

//constexpr int rounding=8;

//GPU auxiliary functions
static inline void check(hipError_t err, const char* context) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << context << ": "
            << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

#define CHECK(x) check(x, #x)

static inline int divup(int a, int b) {
    return (a + b - 1)/b;
}

static inline int roundup(int a, int b) {
    return divup(a, b) * b;
}

//GPU padding kernel
__global__ void paddingkernel(float* copy_x, float* divisions, float* x, float* result, int nx, int nn_x, int ny) {
	//Get thread info
	int ja = threadIdx.x;
	int row = blockIdx.y;
	
	//Transfer padded values to x
    	for (int jb = 0; jb < nn_x; jb+=64) {
		int column=ja+jb;
		float value=0.0;
		if(row<ny && column < nx){
			value=copy_x[row*nx + column]/sqrt(divisions[row]);
		}
		x[row*nn_x + column] = value;
    	}
	//Initialize result
	for(int column=0;column<ny;++column){
		if(row<ny){
			result[row*ny+column]=0.0;
		}
	}
}

//GPU main kernel 

__global__ void mykernel(float* x,float* result, int nx, int nn_x, int ny, int nn_y) {
	int ia = threadIdx.x; 
    	int ja = threadIdx.y; 
    	int ic = blockIdx.x; 
    	int jc = blockIdx.y;

	__shared__ float firsts[4][64];
    	__shared__ float seconds[4][64]; 

	//Initialization
	float pearson_coeff[8][8];
    	for (int ib = 0; ib < 8; ++ib) {
        	for (int jb = 0; jb < 8; ++jb) {
            		pearson_coeff[ib][jb] = 0.0;
        	}
    	}
	
	//Do the calculation
	for (int ks = 0; ks < nn_x; ks += 4) {
		int col=ia*8+ja;
            	int row = ic * 64 + col;
		int row_aux = jc * 64 + col;
		for (int f = 0; f < 4; ++f) {
            		int k = ks + f;
            		firsts[f][col] = x[nn_x*row + k];
            		seconds[f][col] = x[nn_x*row_aux + k];
        	}
			
		__syncthreads();
		
        	//Multiplication
		#pragma unroll
        	for (int f = 0; f < 4; ++f) {
            		float second[8]={0.0,0.0,0.0,0.0,0.0,0.0,0.0,0.0};
			for (int jb = 0; jb < 8; ++jb) {
                		second[jb] = seconds[f][jb * 8 + ja];
            		}
            		for (int ib = 0; ib < 8; ++ib) {
                		float first = firsts[f][ib * 8 + ia];
                		for (int jb = 0; jb < 8; ++jb) {
                			pearson_coeff[ib][jb] += first*second[jb];
            			}
        		}
		}
	
		 __syncthreads();

	}
	for (int ib = 0; ib < 8; ++ib) {
        	for (int jb = 0; jb < 8; ++jb) {
         		int row = ic * 64 + ib * 8 + ia;
            		int row_aux = jc * 64 + jb * 8 + ja;
			if (row < ny && row_aux < ny) {
                		result[ny*row_aux + row] += pearson_coeff[ib][jb];
				//result[ny*row_aux + row] += pearson_coeff[ib][jb];
            		}
        	}
    	}
	
}

//CPU auxiliary functions
//Struct for sum
struct Sum{
	auto operator() (float n){ sum +=n; }
	float sum{0};
};


//Struc for square_sum
struct Square_sum{
	auto operator() (float n){ sq_sum += pow(n,2.0f); }
	float sq_sum{0};
};

void correlate(int ny, int nx, const float *data, float *result) {
	//asm("CPU PART I");
	//Auxiliary variables
	std::vector <float> x(ny*nx,0.0); //x matrix
	std::vector <float> divisions(ny,0.0); //Store the divisions we are going to do in cuda

	//Now x will cointain data mean and std normalized
	for(int row=0;row<ny;++row){
		for (int column=0;column<nx;++column){
			x[column + row*nx]=data[column + row*nx];
		}
		//Mean
		//Sum all values
		Sum s = std::for_each(x.begin()+(row*nx),x.begin()+(nx+row*nx),Sum());
		//Substract mean to all of them
		std::for_each(x.begin()+(row*nx),x.begin()+(nx+row*nx),[&] (float &n){n=n-(s.sum/nx);});

		//Norm
		Square_sum sq_s = std::for_each(x.begin()+(row*nx),x.begin()+(nx+row*nx),Square_sum());
		//Store division
		//std::for_each(x.begin()+(row*nx),x.begin()+(nx+row*nx),[&] (float &n){n=n/sqrt(sq_s.sq_sum);});
		divisions[row]=sq_s.sq_sum;
	}

	//asm("GPU PART");
	int nn_x = roundup(nx, 64);
	int nn_y = roundup(ny, 64);
	//Memory storing
	float* xGPU=NULL; //This one will be padded
	CHECK(hipMalloc((void**)&xGPU, nn_y * nn_x * sizeof(float)));
	
	float* copy_xGPU = NULL; //This one will get the data
    	CHECK(hipMalloc((void**)&copy_xGPU, ny * nx * sizeof(float)));
    	CHECK(hipMemcpy(copy_xGPU, &*x.begin(), ny * nx * sizeof(float), hipMemcpyHostToDevice));
	
	float* copy_divisions = NULL; //This one will get the divisions for each row
    	CHECK(hipMalloc((void**)&copy_divisions, ny * sizeof(float)));
    	CHECK(hipMemcpy(copy_divisions, &*divisions.begin(), ny * sizeof(float), hipMemcpyHostToDevice));
		
	//Preparations for calulation
	float* resultGPU = NULL;
    	CHECK(hipMalloc((void**)&resultGPU, ny * ny * sizeof(float)));
	// Divisions, padding and initialization
    	{
        	dim3 dimBlock(64, 1);
        	dim3 dimGrid(1, nn_y);
        	paddingkernel<<<dimGrid, dimBlock>>>(copy_xGPU, copy_divisions,xGPU, resultGPU, nx, nn_x, ny);
        	CHECK(hipGetLastError());
    	}

	
    	// Run calculations kernel
	{
		dim3 dimBlock(8, 8);
        	dim3 dimGrid(nn_y / 64, nn_y / 64);
    		mykernel<<<dimGrid, dimBlock>>>(xGPU, resultGPU, nx, nn_x, ny, nn_y);
    		CHECK(hipGetLastError());
	}

    	// Copy data back to CPU & release memory
    	CHECK(hipMemcpy(result, resultGPU, ny * ny * sizeof(float), hipMemcpyDeviceToHost));
    	CHECK(hipFree(copy_divisions));
	CHECK(hipFree(copy_xGPU));
	CHECK(hipFree(xGPU));
    	CHECK(hipFree(resultGPU));
}

